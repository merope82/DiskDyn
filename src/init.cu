#include <stdio.h>
#include "ddyn.cuh"

Grid::Grid(){						// Initialization
    host.x     = NULL;
    host.y     = NULL;
    host.z     = NULL;
    host.vx    = NULL;
    host.vy    = NULL;
    host.vz    = NULL;
    host.m     = NULL;
    host.N     = NULL;
    host.id    = NULL;

    host.k1x   = NULL;
    host.k2x   = NULL;
    host.k3x   = NULL;
    host.k4x   = NULL;
    host.k1y   = NULL;
    host.k2y   = NULL;
    host.k3y   = NULL;
    host.k4y   = NULL;
    host.k1z   = NULL;
    host.k2z   = NULL;
    host.k3z   = NULL;
    host.k4z   = NULL;

    host.k2rx  = NULL;
    host.k3rx  = NULL;
    host.k4rx  = NULL;
    host.k2ry  = NULL;
    host.k3ry  = NULL;
    host.k4ry  = NULL;
    host.k2rz  = NULL;
    host.k3rz  = NULL;
    host.k4rz  = NULL;

    host.k1vx  = NULL;
    host.k2vx  = NULL;
    host.k3vx  = NULL;
    host.k4vx  = NULL;
    host.k1vy  = NULL;
    host.k2vy  = NULL;
    host.k3vy  = NULL;
    host.k4vy  = NULL;
    host.k1vz  = NULL;
    host.k2vz  = NULL;
    host.k3vz  = NULL;
    host.k4vz  = NULL;

    gpu.x      = NULL;					// GPU memory pointer
    gpu.y      = NULL;
    gpu.z      = NULL;
    gpu.vx     = NULL;
    gpu.vy     = NULL;
    gpu.vz     = NULL;
    gpu.m      = NULL;
    gpu.N      = NULL;
    gpu.id     = NULL;

    gpu.k1x    = NULL;
    gpu.k2x    = NULL;
    gpu.k3x    = NULL;
    gpu.k4x    = NULL;
    gpu.k1y    = NULL;
    gpu.k2y    = NULL;
    gpu.k3y    = NULL;
    gpu.k4y    = NULL;
    gpu.k1z    = NULL;
    gpu.k2z    = NULL;
    gpu.k3z    = NULL;
    gpu.k4z    = NULL;

    gpu.k2rx   = NULL;
    gpu.k3rx   = NULL;
    gpu.k4rx   = NULL;
    gpu.k2ry   = NULL;
    gpu.k3ry   = NULL;
    gpu.k4ry   = NULL;
    gpu.k2rz   = NULL;
    gpu.k3rz   = NULL;
    gpu.k4rz   = NULL;

    gpu.k1vx   = NULL;
    gpu.k2vx   = NULL;
    gpu.k3vx   = NULL;
    gpu.k4vx   = NULL;
    gpu.k1vy   = NULL;
    gpu.k2vy   = NULL;
    gpu.k3vy   = NULL;
    gpu.k4vy   = NULL;
    gpu.k1vz   = NULL;
    gpu.k2vz   = NULL;
    gpu.k3vz   = NULL;
    gpu.k4vz   = NULL;

    Th         = NULL;
    Td         = NULL;
    w          = NULL;
    in	       = NULL;
    Qabs       = NULL;
    Qsca       = NULL;
    Qpfc       = NULL;
    Fh	       = NULL;
    Fd         = NULL;
    Fdust      = NULL;

    synced     = false;					// Not synced at this point
}

Grid::~Grid(){

    free(host.x);
    free(host.y);
    free(host.z);
    free(host.vx);
    free(host.vy);
    free(host.vz);
    free(host.m);
    free(host.N);
    free(host.id);

    free(host.k1x);
    free(host.k2x);
    free(host.k3x);
    free(host.k4x);
    free(host.k1y);
    free(host.k2y);
    free(host.k3y);
    free(host.k4y);
    free(host.k1z);
    free(host.k2z);
    free(host.k3z);
    free(host.k4z);

    free(host.k2rx);
    free(host.k3rx);
    free(host.k4rx);
    free(host.k2ry);
    free(host.k3ry);
    free(host.k4ry);
    free(host.k2rz);
    free(host.k3rz);
    free(host.k4rz);

    free(host.k1vx);
    free(host.k2vx);
    free(host.k3vx);
    free(host.k4vx);
    free(host.k1vy);
    free(host.k2vy);
    free(host.k3vy);
    free(host.k4vy);
    free(host.k1vz);
    free(host.k2vz);
    free(host.k3vz);
    free(host.k4vz);
    free(Th);
    free(Fh);

    gpuErrchk(hipFree(gpu.x));
    gpuErrchk(hipFree(gpu.y));
    gpuErrchk(hipFree(gpu.z));
    gpuErrchk(hipFree(gpu.vx));
    gpuErrchk(hipFree(gpu.vy));
    gpuErrchk(hipFree(gpu.vz));
    gpuErrchk(hipFree(gpu.m));
    gpuErrchk(hipFree(gpu.N));
    gpuErrchk(hipFree(gpu.id));

    gpuErrchk(hipFree(gpu.k1x));
    gpuErrchk(hipFree(gpu.k2x));
    gpuErrchk(hipFree(gpu.k3x));
    gpuErrchk(hipFree(gpu.k4x));
    gpuErrchk(hipFree(gpu.k1y));
    gpuErrchk(hipFree(gpu.k2y));
    gpuErrchk(hipFree(gpu.k3y));
    gpuErrchk(hipFree(gpu.k4y));
    gpuErrchk(hipFree(gpu.k1z));
    gpuErrchk(hipFree(gpu.k2z));
    gpuErrchk(hipFree(gpu.k3z));
    gpuErrchk(hipFree(gpu.k4z));

    gpuErrchk(hipFree(gpu.k2rx));
    gpuErrchk(hipFree(gpu.k3rx));
    gpuErrchk(hipFree(gpu.k4rx));
    gpuErrchk(hipFree(gpu.k2ry));
    gpuErrchk(hipFree(gpu.k3ry));
    gpuErrchk(hipFree(gpu.k4ry));
    gpuErrchk(hipFree(gpu.k2rz));
    gpuErrchk(hipFree(gpu.k3rz));
    gpuErrchk(hipFree(gpu.k4rz));

    gpuErrchk(hipFree(gpu.k1vx));
    gpuErrchk(hipFree(gpu.k2vx));
    gpuErrchk(hipFree(gpu.k3vx));
    gpuErrchk(hipFree(gpu.k4vx));
    gpuErrchk(hipFree(gpu.k1vy));
    gpuErrchk(hipFree(gpu.k2vy));
    gpuErrchk(hipFree(gpu.k3vy));
    gpuErrchk(hipFree(gpu.k4vy));
    gpuErrchk(hipFree(gpu.k1vz));
    gpuErrchk(hipFree(gpu.k2vz));
    gpuErrchk(hipFree(gpu.k3vz));
    gpuErrchk(hipFree(gpu.k4vz));

    gpuErrchk(hipFree(w));
    gpuErrchk(hipFree(in));
    gpuErrchk(hipFree(Td));
    gpuErrchk(hipFree(Qabs));
    gpuErrchk(hipFree(Qsca));
    gpuErrchk(hipFree(Qpfc));
    gpuErrchk(hipFree(Fd));
    gpuErrchk(hipFree(Fdust));
}

void Grid::allochost(){
    configdata *cfg  = &cconfig;

    if (!host.x   )  host.x      = (double *)malloc(sizeof(double)*n_all);
	if ( !host.x )		 exit_with_usage(53);
    if (!host.y   )  host.y      = (double *)malloc(sizeof(double)*n_all);
	if ( !host.y )		 exit_with_usage(53);
    if (!host.z   )  host.z      = (double *)malloc(sizeof(double)*n_all);
	if ( !host.z )		 exit_with_usage(53);
    if (!host.vx  )  host.vx     = (double *)malloc(sizeof(double)*n_all);
	if ( !host.vx )		 exit_with_usage(53);
    if (!host.vy  )  host.vy     = (double *)malloc(sizeof(double)*n_all);
	if ( !host.vy )		 exit_with_usage(53);
    if (!host.vz  )  host.vz     = (double *)malloc(sizeof(double)*n_all);
	if ( !host.vz )	         exit_with_usage(53);
    if (!host.m   )  host.m      = (double *)malloc(sizeof(double)*n_all);
	if ( !host.m )		 exit_with_usage(53);
    if (!host.N   )  host.N      = (double *)malloc(sizeof(double)*n_all);
	if ( !host.N )		 exit_with_usage(53);
    if (!host.id  )  host.id     = (int    *)malloc(sizeof(int)   *n_all);
	if ( !host.id )		 exit_with_usage(53);
    if (!Fh       )  Fh          = (double *)malloc(sizeof(double)*cfg->xsize*cfg->ysize);
	if ( !Fh )		 exit_with_usage(53);

    if (!host.k1x )  host.k1x    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k1x )	 exit_with_usage(53);
    if (!host.k2x )  host.k2x    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k2x )	 exit_with_usage(53);
    if (!host.k3x )  host.k3x    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k3x )	 exit_with_usage(53);
    if (!host.k4x )  host.k4x    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k4x )	 exit_with_usage(53);
    if (!host.k1y )  host.k1y    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k1y )	 exit_with_usage(53);
    if (!host.k2y )  host.k2y    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k2y )	 exit_with_usage(53);
    if (!host.k3y )  host.k3y    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k3y )	 exit_with_usage(53);
    if (!host.k4y )  host.k4y    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k4y )	 exit_with_usage(53);
    if (!host.k1z )  host.k1z    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k1z )	 exit_with_usage(53);
    if (!host.k2z )  host.k2z    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k2z )	 exit_with_usage(53);
    if (!host.k3z )  host.k3z    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k3z )	 exit_with_usage(53);
    if (!host.k4z )  host.k4z    = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k4z )	 exit_with_usage(53);

    if (!host.k2rx)  host.k2rx   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k2rx )	 exit_with_usage(53);
    if (!host.k3rx)  host.k3rx   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k3rx )	 exit_with_usage(53);
    if (!host.k4rx)  host.k4rx   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k4rx )	 exit_with_usage(53);
    if (!host.k2ry)  host.k2ry   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k2ry )	 exit_with_usage(53);
    if (!host.k3ry)  host.k3ry   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k3ry )	 exit_with_usage(53);
    if (!host.k4ry)  host.k4ry   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k4ry )	 exit_with_usage(53);
    if (!host.k2rz)  host.k2rz   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k2rz )	 exit_with_usage(53);
    if (!host.k3rz)  host.k3rz   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k3rz )	 exit_with_usage(53);
    if (!host.k4rz)  host.k4rz   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k4rz )	 exit_with_usage(53);

    if (!host.k1vx)  host.k1vx   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k1vx )	 exit_with_usage(53);
    if (!host.k2vx)  host.k2vx   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k2vx )	 exit_with_usage(53);
    if (!host.k3vx)  host.k3vx   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k3vx )	 exit_with_usage(53);
    if (!host.k4vx)  host.k4vx   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k4vx )	 exit_with_usage(53);
    if (!host.k1vy)  host.k1vy   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k1vy )	 exit_with_usage(53);
    if (!host.k2vy)  host.k2vy   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k2vy )	 exit_with_usage(53);
    if (!host.k3vy)  host.k3vy   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k3vy )	 exit_with_usage(53);
    if (!host.k4vy)  host.k4vy   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k4vy )	 exit_with_usage(53);
    if (!host.k1vz)  host.k1vz   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k1vz )	 exit_with_usage(53);
    if (!host.k2vz)  host.k2vz   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k2vz )	 exit_with_usage(53);
    if (!host.k3vz)  host.k3vz   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k3vz )	 exit_with_usage(53);
    if (!host.k4vz)  host.k4vz   = (double *)malloc(sizeof(double)*n_all);
	if ( !host.k4vz )	 exit_with_usage(53);

    cfg->mem +=(sizeof(double)*41+sizeof(int))*n_all+sizeof(double)*cfg->xsize*cfg->ysize;

    null_host();
}

void Grid::reallochost(int plus){
    configdata *cfg  = &cconfig;
    double *dtmp;
    int	   *itmp;

    dtmp        = (double *)realloc(host.x   ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.x = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.y   ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.y = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.z   ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.z = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.vx   ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.vx = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.vy   ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.vy = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.vz   ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.vz = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.m   ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.m = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.N   ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.N = dtmp;
    else	exit_with_usage(53);

    itmp        = (int *)realloc(host.id     ,sizeof(int)*(n_all+plus));
    if (itmp)	host.id = itmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k1x ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k1x = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k2x ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k2x = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k3x ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k3x = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k4x ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k4x = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k1y ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k1y = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k2y ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k2y = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k3y ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k3y = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k4y ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k4y = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k1z ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k1z = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k2z ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k2z = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k3z ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k3z = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k4z ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k4z = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k2rx ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k2rx = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k3rx ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k3rx = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k4rx ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k4rx = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k2ry ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k2ry = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k3ry ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k3ry = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k4ry ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k4ry = dtmp;
    else	exit_with_usage(53);
                                    
    dtmp        = (double *)realloc(host.k2rz ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k2rz = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k3rz ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k3rz = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k4rz ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k4rz = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k1vx ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k1vx = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k2vx ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k2vx = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k3vx ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k3vx = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k4vx ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k4vx = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k1vy ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k1vy = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k2vy ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k2vy = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k3vy ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k3vy = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k4vy ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k4vy = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k1vz ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k1vz = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k2vz ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k2vz = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k3vz ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k3vz = dtmp;
    else	exit_with_usage(53);

    dtmp        = (double *)realloc(host.k4vz ,sizeof(double)*(n_all+plus));
    if (dtmp)	host.k4vz = dtmp;
    else	exit_with_usage(53);

    cfg->mem += (sizeof(double)*41+sizeof(int))*plus;
}

void Grid::null_host(){

    for ( int j=0 ; j<n_all ; j++ ){
        host.x[j]    = 0;
	host.y[j]    = 0;
	host.z[j]    = 0;
	host.vx[j]   = 0;
	host.vy[j]   = 0;
	host.vz[j]   = 0;
	host.m[j]    = 0;
        host.N[j]    = 0;
        host.id[j]   = 0;
        
        host.k1x[j]  = 0;
        host.k2x[j]  = 0;
        host.k3x[j]  = 0;
        host.k4x[j]  = 0;
        host.k1y[j]  = 0;
        host.k2y[j]  = 0;
        host.k3y[j]  = 0;
        host.k4y[j]  = 0;
        host.k1z[j]  = 0;
        host.k2z[j]  = 0;
        host.k3z[j]  = 0;
        host.k4z[j]  = 0;
        
        host.k2rx[j] = 0;
        host.k3rx[j] = 0;
        host.k4rx[j] = 0;
        host.k2ry[j] = 0;
        host.k3ry[j] = 0;
        host.k4ry[j] = 0;
        host.k2rz[j] = 0;
        host.k3rz[j] = 0;
        host.k4rz[j] = 0;
        
        host.k1vx[j] = 0;
        host.k2vx[j] = 0;
        host.k3vx[j] = 0;
        host.k4vx[j] = 0;
        host.k1vy[j] = 0;
        host.k2vy[j] = 0;
        host.k3vy[j] = 0;
        host.k4vy[j] = 0;
        host.k1vz[j] = 0;
        host.k2vz[j] = 0;
        host.k3vz[j] = 0;
        host.k4vz[j] = 0;
    }
    synced   = false;
}

void Grid::allocgpu(){
    configdata *cfg  = &cconfig;

    gpuErrchk(hipMalloc(&gpu.x   ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.y   ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.z   ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.vx  ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.vy  ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.vz  ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.m   ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.N   ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.id  ,sizeof(int)   *n_all));

    gpuErrchk(hipMalloc(&gpu.k1x ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k2x ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k3x ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k4x ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k1y ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k2y ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k3y ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k4y ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k1z ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k2z ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k3z ,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k4z ,sizeof(double)*n_all)); //12

    gpuErrchk(hipMalloc(&gpu.k2rx,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k3rx,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k4rx,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k2ry,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k3ry,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k4ry,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k2rz,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k3rz,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k4rz,sizeof(double)*n_all)); //9

    gpuErrchk(hipMalloc(&gpu.k1vx,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k2vx,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k3vx,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k4vx,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k1vy,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k2vy,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k3vy,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k4vy,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k1vz,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k2vz,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k3vz,sizeof(double)*n_all));
    gpuErrchk(hipMalloc(&gpu.k4vz,sizeof(double)*n_all)); //12 - 41 double 1 int,

    gpuErrchk(hipMalloc(&w    ,sizeof(double)*cfg->n_wav));
    gpuErrchk(hipMalloc(&in   ,sizeof(double)*cfg->n_wav));
    gpuErrchk(hipMalloc(&Td   ,sizeof(double)*cfg->n_dust*nR));
    gpuErrchk(hipMalloc(&Fd   ,sizeof(double)*cfg->xsize*cfg->ysize));
    gpuErrchk(hipMalloc(&Qabs ,sizeof(double)*cfg->n_dust*cfg->n_wav));
    gpuErrchk(hipMalloc(&Qsca ,sizeof(double)*cfg->n_dust*cfg->n_wav));
    gpuErrchk(hipMalloc(&Qpfc ,sizeof(double)*cfg->n_dust*cfg->n_wav*cfg->n_theta));
    gpuErrchk(hipMalloc(&Fdust,sizeof(double)*n_dust));

    synced   = false;
}


void Grid::add_to_host(double x,double y,double z,double vx,double vy,double vz,\
                         double N,double m,int id,int tot){

    print_status(n_all,tot-1);
    host.x[n_all]    = x;
    host.y[n_all]    = y;
    host.z[n_all]    = z;
    host.vx[n_all]   = vx;
    host.vy[n_all]   = vy;
    host.vz[n_all]   = vz;
    host.id[n_all]   = id;
    host.m[n_all]    = m;
    host.N[n_all]    = N;
    n_all++;

}

void Grid::sync_to_gpu(){
    if (!synced ){

    gpuErrchk(hipMemcpy(gpu.x   ,host.x   ,sizeof(double)*n_all,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu.y   ,host.y   ,sizeof(double)*n_all,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu.z   ,host.z   ,sizeof(double)*n_all,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu.vx  ,host.vx  ,sizeof(double)*n_all,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu.vy  ,host.vy  ,sizeof(double)*n_all,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu.vz  ,host.vz  ,sizeof(double)*n_all,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu.m   ,host.m   ,sizeof(double)*n_all,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu.N   ,host.N   ,sizeof(double)*n_all,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu.id  ,host.id  ,sizeof(int)   *n_all,hipMemcpyHostToDevice));

    synced = true;
    }
}

void Grid::sync_dust_to_gpu(){
    configdata *cfg  = &cconfig;

    gpuErrchk(hipMemcpy(Qabs    ,cfg->Qabs       ,sizeof(double)*cfg->n_dust*cfg->n_wav,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(Qsca    ,cfg->Qsca       ,sizeof(double)*cfg->n_dust*cfg->n_wav,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(Qpfc    ,cfg->Qpfunc     ,sizeof(double)*cfg->n_dust*cfg->n_wav*cfg->n_theta,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(Td      ,Th              ,sizeof(double)*cfg->n_dust*nR,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(w       ,cfg->w          ,sizeof(double)*cfg->n_wav,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(in      ,cfg->in         ,sizeof(double)*cfg->n_wav,hipMemcpyHostToDevice));

    double *psize;
    float *bvalue;
    float *bswvalue;
    float *qvalue;
    hipGetSymbolAddress((void **)&psize,size); // get a pointer to size
    hipMemcpy(psize,cfg->s,sizeof(double)*cfg->n_dust,hipMemcpyHostToDevice);
    hipGetSymbolAddress((void **)&bvalue,beta); // get a pointer to beta 
    hipMemcpy(bvalue,cfg->beta,sizeof(float)*cfg->n_dust,hipMemcpyHostToDevice);
    hipGetSymbolAddress((void **)&bswvalue,betasw); // get a pointer to betasw 
    hipMemcpy(bswvalue,cfg->betasw,sizeof(float)*cfg->n_dust,hipMemcpyHostToDevice);
    hipGetSymbolAddress((void **)&qvalue,q); // get a pointer to size
    hipMemcpy(qvalue,cfg->q,sizeof(float)*cfg->n_dust,hipMemcpyHostToDevice);

//    hipMemcpyToSymbol(HIP_SYMBOL(size),cfg->s,sizeof(double)*cfg->n_dust,0,hipMemcpyHostToDevice);
//    hipMemcpyToSymbol(HIP_SYMBOL(beta),cfg->beta,sizeof(float)*cfg->n_dust,0,hipMemcpyHostToDevice);
//    hipMemcpyToSymbol(HIP_SYMBOL(betasw),cfg->betasw,sizeof(float)*cfg->n_dust,0,hipMemcpyHostToDevice);
//    hipMemcpyToSymbol(HIP_SYMBOL(q),cfg->q,sizeof(float)*cfg->n_dust,0,hipMemcpyHostToDevice);

}


void Grid::sync_from_gpu(){
    if (!synced ){

    gpuErrchk(hipMemcpy(host.x   ,gpu.x   ,sizeof(double)*n_all,hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(host.y   ,gpu.y   ,sizeof(double)*n_all,hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(host.z   ,gpu.z   ,sizeof(double)*n_all,hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(host.vx  ,gpu.vx  ,sizeof(double)*n_all,hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(host.vy  ,gpu.vy  ,sizeof(double)*n_all,hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(host.vz  ,gpu.vz  ,sizeof(double)*n_all,hipMemcpyDeviceToHost));

    synced = true;
    }
}
